#include "hip/hip_runtime.h"
#include <torch/serialize/tensor.h>
#include <THC/THC.h>

#include "./../gpu.h"
#include "./defines.h"
#include "./pairwise_ops.h"


namespace prob_phoc {
namespace gpu {

template <typename T, typename O>
__global__
void cphoc_kernel(const long int na, const long int nb, const long int d, const T* xa, const T* xb, T* y, const O op) {
  for (long int i = thGy; i < na; i += NTGy) {
    for (long int j = thGx; j < nb; j += NTGx) {
      const T* xa_i = xa + i * d;
      const T* xb_j = xb + j * d;
      y[i * nb + j] = op(d, xa_i, xb_j);
    }
  }
}

template <typename T, typename O>
__global__
void pphoc_kernel(const long int n, const long int d, const T* x, T* y, const O op) {
  for (long int i = thGy; i < n; i += NTGy) {
    for (long int j = thGx; j < n; j += NTGx) {
      if (j > i) {
        const T* x_i = x + i * d;
        const T* x_j = x + j * d;
        const long k = i * (2 * n - i - 1) / 2 + (j - i - 1);
        y[k] = op(d, x_i, x_j);
      }
    }
  }
}

template <typename T, typename O>
void Impl<T, O>::cphoc(const c10::Device& device, const long int na, const long int nb, const long int d, const T* xa, const T* xb, T* y) const {
  c10::DeviceGuard device_guard(device);
  auto stream = THCState_getCurrentStream(at::globalContext().getTHCState());
  const dim3 block_size(32, 32);
  const dim3 grid_size(NUM_BLOCKS(na, 32),
                       NUM_BLOCKS(nb, 32));
  cphoc_kernel<T, O><<<grid_size, block_size, 0, stream>>>(na, nb, d, xa, xb, y, op_);
  if (stream == nullptr) {
    CHECK_LAST_CUDA_CALL();
  }
}

template <typename T, typename O>
void Impl<T, O>::pphoc(const c10::Device& device, const long int n, const long int d, const T* x, T* y) const {
  c10::DeviceGuard device_guard(device);
  auto stream = THCState_getCurrentStream(at::globalContext().getTHCState());
  const dim3 block_size(32, 32);
  const dim3 grid_size(NUM_BLOCKS(n, 32),
                       NUM_BLOCKS(n, 32));
  pphoc_kernel<T, O><<<grid_size, block_size, 0, stream>>>(n, d, x, y, op_);
  if (stream == nullptr) {
    CHECK_LAST_CUDA_CALL();
  }
}

template class SumProdLogSemiring<float>;
template class SumProdLogSemiring<double>;

template class SumProdRealSemiring<float>;
template class SumProdRealSemiring<double>;

}  // namespace gpu
}  // namespace prob_phoc
